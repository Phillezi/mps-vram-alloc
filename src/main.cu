#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <vector>
#include <string>
#include <chrono>
#include <hip/hip_runtime.h>
#include "json.hpp"

using json = nlohmann::json;

__global__ void vectorAdd(const float* A, const float* B, float* C, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) C[i] = A[i] + B[i];
}

void checkCuda(hipError_t result, const char* msg) {
    if (result != hipSuccess) {
        std::cerr << "CUDA error: " << msg << " >>> " << hipGetErrorString(result) << "\n";
        exit(EXIT_FAILURE);
    }
}

bool tryAlloc(size_t bytes) {
    void* ptr = nullptr;
    hipError_t err = hipMalloc(&ptr, bytes);
    if (err == hipSuccess) {
        hipFree(ptr);
        return true;
    }
    hipGetLastError();
    return false;
}

size_t findMaxAllocatableVRAM(size_t totalMem) {
    size_t low = 0, high = totalMem, best = 0;
    while (low <= high) {
        size_t mid = (low + high) / 2;
        if (tryAlloc(mid)) {
            best = mid;
            low = mid + (16 * 1024 * 1024);
        } else {
            if (mid < 16 * 1024 * 1024) break;
            high = mid - (16 * 1024 * 1024);
        }
    }
    return best;
}

size_t findTotalUsableVRAM(size_t blockSizeMB) {
    size_t blockSize = blockSizeMB * 1024 * 1024;
    std::vector<void*> blocks;
    size_t totalAllocated = 0;

    while (true) {
        void* ptr = nullptr;
        hipError_t err = hipMalloc(&ptr, blockSize);
        if (err == hipSuccess) {
            blocks.push_back(ptr);
            totalAllocated += blockSize;
        } else if (err == hipErrorOutOfMemory) {
            hipGetLastError();
            break;
        } else {
            std::cerr << "Unexpected error during allocation: " << hipGetErrorString(err) << "\n";
            break;
        }
    }

    for (void* ptr : blocks) hipFree(ptr);
    return totalAllocated;
}

std::string formatDouble(double val, int precision = 1) {
    std::ostringstream oss;
    oss.imbue(std::locale(""));
    oss << std::fixed << std::setprecision(precision) << val;
    return oss.str();
}

int main() {
    int deviceCount = 0;
    checkCuda(hipGetDeviceCount(&deviceCount), "get device count");
    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found.\n";
        return 1;
    }

    std::vector<json> results;

    for (int dev = 0; dev < deviceCount; ++dev) {
        hipDeviceProp_t prop;
        checkCuda(hipGetDeviceProperties(&prop, dev), "get device properties");

        std::cerr << "\n=== Device " << dev << " ===\n";
        std::cerr << "Name: " << prop.name << "\n";
        std::cerr << "Total VRAM: " << formatDouble(prop.totalGlobalMem / (1024.0 * 1024.0)) << " MB\n";
        std::cerr << "Compute Capability: " << prop.major << "." << prop.minor << "\n";

        checkCuda(hipSetDevice(dev), "set device");

        std::cerr << "[Finding max single allocatable chunk...]\n";
        auto t1 = std::chrono::high_resolution_clock::now();
        size_t maxAlloc = findMaxAllocatableVRAM(prop.totalGlobalMem);
        auto t2 = std::chrono::high_resolution_clock::now();
        double allocTime = std::chrono::duration<double, std::milli>(t2 - t1).count();
        std::cerr << "Max single alloc: " << formatDouble(maxAlloc / (1024.0 * 1024.0)) << " MB\n";

        std::cerr << "[Finding total usable VRAM using multiple allocations...]\n";
        auto t3 = std::chrono::high_resolution_clock::now();
        size_t totalUsable = findTotalUsableVRAM(128);
        auto t4 = std::chrono::high_resolution_clock::now();
        double totalAllocTime = std::chrono::duration<double, std::milli>(t4 - t3).count();
        std::cerr << "Total usable: " << formatDouble(totalUsable / (1024.0 * 1024.0)) << " MB\n";

        int N = 1 << 24;
        size_t bytes = N * sizeof(float);
        float *A, *B, *C;
        checkCuda(hipMalloc(&A, bytes), "alloc A");
        checkCuda(hipMalloc(&B, bytes), "alloc B");
        checkCuda(hipMalloc(&C, bytes), "alloc C");

        int blockSize = 256;
        int gridSize = (N + blockSize - 1) / blockSize;

        auto start = std::chrono::high_resolution_clock::now();
        vectorAdd<<<gridSize, blockSize>>>(A, B, C, N);
        hipDeviceSynchronize();
        auto end = std::chrono::high_resolution_clock::now();

        double ms = std::chrono::duration<double, std::milli>(end - start).count();
        double gbTransferred = 3.0 * bytes / (1024.0 * 1024.0 * 1024.0);
        double bandwidth = gbTransferred / (ms / 1000.0);

        hipFree(A);
        hipFree(B);
        hipFree(C);

        json entry = {
            {"device_id", dev},
            {"name", prop.name},
            {"compute_capability", std::to_string(prop.major) + "." + std::to_string(prop.minor)},
            {"total_vram_mb", prop.totalGlobalMem / (1024.0 * 1024.0)},
            {"max_single_alloc_mb", maxAlloc / (1024.0 * 1024.0)},
            {"alloc_test_time_ms", allocTime},
            {"total_usable_vram_mb", totalUsable / (1024.0 * 1024.0)},
            {"multi_alloc_time_ms", totalAllocTime},
            {"vector_add_time_ms", ms},
            {"bandwidth_gbps", bandwidth}
        };

        results.push_back(entry);
    }

    std::cerr << "\n================ CUDA BENCHMARK SUMMARY ================\n";
    std::cerr << std::left << std::setw(25) << "Device"
              << std::setw(12) << "VRAM(MB)"
              << std::setw(15) << "SingleAlloc"
              << std::setw(15) << "TotalUsable"
              << std::setw(15) << "BW(GB/s)"
              << "\n--------------------------------------------------------\n";

    for (auto& r : results) {
        std::cerr << std::left
                  << std::setw(25) << r["name"].get<std::string>()
                  << std::setw(12) << formatDouble(r["total_vram_mb"].get<double>())
                  << std::setw(15) << formatDouble(r["max_single_alloc_mb"].get<double>())
                  << std::setw(15) << formatDouble(r["total_usable_vram_mb"].get<double>())
                  << std::setw(15) << formatDouble(r["bandwidth_gbps"].get<double>())
                  << "\n";
    }

    std::string file = "cuda_benchmark_results.json";
    std::ifstream ifs(file);
    json existing = json::array();
    if (ifs) { try { ifs >> existing; } catch (...) {} }
    ifs.close();

    for (auto& r : results) existing.push_back(r);
    std::ofstream ofs(file);
    ofs << std::setw(4) << existing;
    ofs.close();

    std::cerr << "\nResults appended to " << file << "\n";
    return 0;
}
